#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include "stb_image_write.h"

__global__ void render(float* fb, int max_x, int max_y)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int pixel_index = j * max_x * 3 + i * 3;

	fb[pixel_index + 0] = float(i) / max_x;
	fb[pixel_index + 1] = float(j) / max_y;
	fb[pixel_index + 2] = 0.2;
}

int main()
{
	int image_width = 2560;
	int image_height = 1440;
	int thread_x = 8;
	int thread_y = 8;

	int num_pixels = image_width * image_height;
	size_t fb_size = 3 * num_pixels * sizeof(float);

	float* fb;
	hipMallocManaged((void**)&fb, fb_size);

	auto start = std::chrono::high_resolution_clock::now();

	dim3 blocks(image_width / thread_x, image_height / thread_y);
	dim3 threads(thread_x, thread_y);

	render<<<blocks, threads>>>(fb, image_width, image_height);

	hipGetLastError();
	hipDeviceSynchronize();

	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
	std::cout << "\nTook " << duration.count() << " milliseconds" << std::endl;

	std::vector<uint8_t> image_data;

	for (int j = image_height - 1; j >= 0; --j)
	{
		for (int i = 0; i < image_width; ++i)
		{
			size_t pixel_index = j * 3 * image_width + i * 3;
			float r = fb[pixel_index + 0];
			float g = fb[pixel_index + 1];
			float b = fb[pixel_index + 2];

			int ir = static_cast<int>(255.999 * r);
			int ig = static_cast<int>(255.999 * g);
			int ib = static_cast<int>(255.999 * b);

			image_data.push_back(ir);
			image_data.push_back(ig);
			image_data.push_back(ib);
		}
	}

	stbi_write_png("Amethyst.png", image_width, image_height, 3, image_data.data(), image_width * 3);

	hipFree(fb);
}